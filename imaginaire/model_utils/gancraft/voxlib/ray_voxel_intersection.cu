#include "hip/hip_runtime.h"
// Copyright (C) 2021 NVIDIA CORPORATION & AFFILIATES.  All rights reserved.
//
// This work is made available under the Nvidia Source Code License-NC.
// To view a copy of this license, check out LICENSE.md
//
// The ray marching algorithm used in this file is a variety of modified Bresenham method:
// http://citeseerx.ist.psu.edu/viewdoc/download?doi=10.1.1.42.3443&rep=rep1&type=pdf
// Search for "voxel traversal algorithm" for related information

#include <torch/types.h>

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

//#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <vector>

#include "voxlib_common.h"

struct RVIP_Params {
    int voxel_dims[3];
    int voxel_strides[3];
    int max_samples;
    int img_dims[2];
    // Camera parameters
    float cam_ori[3];
    float cam_fwd[3];
    float cam_side[3];
    float cam_up[3];
    float cam_c[2];
    float cam_f;
    //unsigned long seed;
};

/*
    out_voxel_id: torch CUDA int32  [   img_dims[0], img_dims[1], max_samples, 1]
    out_depth:    torch CUDA float  [2, img_dims[0], img_dims[1], max_samples, 1]
    out_raydirs:  torch CUDA float  [   img_dims[0], img_dims[1],           1, 3]
    Image coordinates refer to the center of the pixel
    [0, 0, 0] at voxel coordinate is at the corner of the corner block (instead of at the center)
*/
template <int TILE_DIM>
static __global__ void ray_voxel_intersection_perspective_kernel(int32_t* __restrict__ out_voxel_id, float* __restrict__ out_depth, float* __restrict__ out_raydirs,
const int32_t* __restrict__ in_voxel, const RVIP_Params p) {

    int img_coords[2];
    img_coords[1] = blockIdx.x*TILE_DIM+threadIdx.x;
    img_coords[0] = blockIdx.y*TILE_DIM+threadIdx.y;
    if (img_coords[0] >= p.img_dims[0] || img_coords[1] >= p.img_dims[1]) {
        return;
    }
    int pix_index = img_coords[0] * p.img_dims[1] + img_coords[1];

    // Calculate ray origin and direction
    float rayori[3], raydir[3];
    rayori[0] = p.cam_ori[0];
    rayori[1] = p.cam_ori[1];
    rayori[2] = p.cam_ori[2];

    // Camera intrinsics
    float ndc_imcoords[2];
    ndc_imcoords[0] = p.cam_c[0] - (float)img_coords[0]; // Flip height
    ndc_imcoords[1] = (float)img_coords[1] - p.cam_c[1];

    raydir[0] = p.cam_up[0] * ndc_imcoords[0] + p.cam_side[0] * ndc_imcoords[1] + p.cam_fwd[0] * p.cam_f;
    raydir[1] = p.cam_up[1] * ndc_imcoords[0] + p.cam_side[1] * ndc_imcoords[1] + p.cam_fwd[1] * p.cam_f;
    raydir[2] = p.cam_up[2] * ndc_imcoords[0] + p.cam_side[2] * ndc_imcoords[1] + p.cam_fwd[2] * p.cam_f;
    normalize<float, 3>(raydir);

    // Save out_raydirs
    out_raydirs[pix_index*3] = raydir[0];
    out_raydirs[pix_index*3+1] = raydir[1];
    out_raydirs[pix_index*3+2] = raydir[2];

    float axis_t[3];
    int axis_int[3];
    //int axis_intbound[3];

    // Current voxel
    axis_int[0] = floorf(rayori[0]);
    axis_int[1] = floorf(rayori[1]);
    axis_int[2] = floorf(rayori[2]);

    #pragma unroll
    for (int i=0; i<3; i++) {
        if (raydir[i] > 0) {
            // Initial t value
            // Handle boundary case where rayori[i] is a whole number. Always round Up for the next block
            //axis_t[i] = (ceilf(nextafterf(rayori[i], HUGE_VALF)) - rayori[i]) / raydir[i];
            axis_t[i] = ((float)(axis_int[i]+1) - rayori[i]) / raydir[i];
        } else if (raydir[i] < 0) {
            axis_t[i] = ((float)axis_int[i] - rayori[i]) / raydir[i];
        } else {
            axis_t[i] = HUGE_VALF;
        }
    }

    // Fused raymarching and sampling
    bool quit = false;
    for (int cur_plane=0; cur_plane < p.max_samples; cur_plane++) { // Last cycle is for calculating p2
        float t = nanf("0");
        float t2 = nanf("0");
        int32_t blk_id = 0;
        // Find the next intersection
        while (!quit) {
            // Find the next smallest t
            float tnow;
            /*
            #pragma unroll
            for (int i=0; i<3; i++) {
                if (axis_t[i] <= axis_t[(i+1)%3] && axis_t[i] <= axis_t[(i+2)%3]) {
                    // Update current t
                    tnow = axis_t[i];
                    // Update t candidates
                    if (raydir[i] > 0) {
                        axis_int[i] += 1;
                        if (axis_int[i] >= p.voxel_dims[i]) {
                            quit = true;
                        }
                        axis_t[i] = ((float)(axis_int[i]+1) - rayori[i]) / raydir[i];
                    } else {
                        axis_int[i] -= 1;
                        if (axis_int[i] < 0) {
                            quit = true;
                        }
                        axis_t[i] = ((float)axis_int[i] - rayori[i]) / raydir[i];
                    }
                    break; // Avoid advancing multiple steps as axis_t is updated
                }
            }
            */
            // Hand unroll
            if (axis_t[0] <= axis_t[1] && axis_t[0] <= axis_t[2]) {
                // Update current t
                tnow = axis_t[0];
                // Update t candidates
                if (raydir[0] > 0) {
                    axis_int[0] += 1;
                    if (axis_int[0] >= p.voxel_dims[0]) {
                        quit = true;
                    }
                    axis_t[0] = ((float)(axis_int[0]+1) - rayori[0]) / raydir[0];
                } else {
                    axis_int[0] -= 1;
                    if (axis_int[0] < 0) {
                        quit = true;
                    }
                    axis_t[0] = ((float)axis_int[0] - rayori[0]) / raydir[0];
                }
            } else if (axis_t[1] <= axis_t[2]) {
                tnow = axis_t[1];
                if (raydir[1] > 0) {
                    axis_int[1] += 1;
                    if (axis_int[1] >= p.voxel_dims[1]) {
                        quit = true;
                    }
                    axis_t[1] = ((float)(axis_int[1]+1) - rayori[1]) / raydir[1];
                } else {
                    axis_int[1] -= 1;
                    if (axis_int[1] < 0) {
                        quit = true;
                    }
                    axis_t[1] = ((float)axis_int[1] - rayori[1]) / raydir[1];
                }
            } else {
                tnow = axis_t[2];
                if (raydir[2] > 0) {
                    axis_int[2] += 1;
                    if (axis_int[2] >= p.voxel_dims[2]) {
                        quit = true;
                    }
                    axis_t[2] = ((float)(axis_int[2]+1) - rayori[2]) / raydir[2];
                } else {
                    axis_int[2] -= 1;
                    if (axis_int[2] < 0) {
                        quit = true;
                    }
                    axis_t[2] = ((float)axis_int[2] - rayori[2]) / raydir[2];
                }
            }

            if (quit) {
                break;
            }

            // Skip empty space
            // Could there be deadlock if the ray direction is away from the world?
            if (axis_int[0] < 0 || axis_int[0] >= p.voxel_dims[0] || axis_int[1] < 0 || axis_int[1] >= p.voxel_dims[1] || axis_int[2] < 0 || axis_int[2] >= p.voxel_dims[2]) {
                continue;
            }

            // Test intersection using voxel grid
            blk_id = in_voxel[axis_int[0]*p.voxel_strides[0] + axis_int[1]*p.voxel_strides[1] + axis_int[2]*p.voxel_strides[2]];
            if (blk_id == 0) {
                continue;
            }

            // Now that there is an intersection
            t = tnow;
            // Calculate t2
            /*
            #pragma unroll
            for (int i=0; i<3; i++) {
                if (axis_t[i] <= axis_t[(i+1)%3] && axis_t[i] <= axis_t[(i+2)%3]) {
                    t2 = axis_t[i];
                    break;
                }
            }
            */
            // Hand unroll
            if (axis_t[0] <= axis_t[1] && axis_t[0] <= axis_t[2]) {
                t2 = axis_t[0];
            } else if (axis_t[1] <= axis_t[2]) {
                t2 = axis_t[1];
            } else {
                t2 = axis_t[2];
            }
            break;
        } // while !quit (ray marching loop)

        out_depth[pix_index*p.max_samples+cur_plane] = t;
        out_depth[p.img_dims[0]*p.img_dims[1]*p.max_samples + pix_index*p.max_samples+cur_plane] = t2;
        out_voxel_id[pix_index*p.max_samples+cur_plane] = blk_id;
    } // cur_plane
}


/*
    out:
        out_voxel_id: torch CUDA int32  [   img_dims[0], img_dims[1], max_samples, 1]
        out_depth:    torch CUDA float  [2, img_dims[0], img_dims[1], max_samples, 1]
        out_raydirs:  torch CUDA float  [   img_dims[0], img_dims[1],           1, 3]
    in:
        in_voxel:     torch CUDA int32  [X, Y, Z] [40, 512, 512]
        cam_ori:      torch      float  [3]
        cam_dir:      torch      float  [3]
        cam_up:       torch      float  [3]
        cam_f:                   float
        cam_c:                   int    [2]
        img_dims:                int    [2]
        max_samples:             int
*/
std::vector<torch::Tensor> ray_voxel_intersection_perspective_cuda(const torch::Tensor& in_voxel, const torch::Tensor& cam_ori, const torch::Tensor& cam_dir, const torch::Tensor& cam_up, float cam_f, const std::vector<float>& cam_c, const std::vector<int>& img_dims, int max_samples) {
    CHECK_CUDA(in_voxel);

    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);
    torch::Device device = in_voxel.device();

    //assert(in_voxel.dtype() == torch::kU8);
    assert(in_voxel.dtype() == torch::kInt32); // Minecraft compatibility
    assert(in_voxel.dim() == 3);
    assert(cam_ori.dtype() == torch::kFloat32);
    assert(cam_ori.numel() == 3);
    assert(cam_dir.dtype() == torch::kFloat32);
    assert(cam_dir.numel() == 3);
    assert(cam_up.dtype() == torch::kFloat32);
    assert(cam_up.numel() == 3);
    assert(img_dims.size() == 2);

    RVIP_Params p;

    // Calculate camera rays
    const torch::Tensor cam_ori_c = cam_ori.cpu();
    const torch::Tensor cam_dir_c = cam_dir.cpu();
    const torch::Tensor cam_up_c = cam_up.cpu();

    // Get the coordinate frame of camera space in world space
    normalize<float, 3>(p.cam_fwd, cam_dir_c.data_ptr<float>());
    cross<float>(p.cam_side, p.cam_fwd, cam_up_c.data_ptr<float>());
    normalize<float, 3>(p.cam_side);
    cross<float>(p.cam_up, p.cam_side, p.cam_fwd);
    normalize<float, 3>(p.cam_up); // Not absolutely necessary as both vectors are normalized. But just in case...

    copyarr<float, 3>(p.cam_ori, cam_ori_c.data_ptr<float>());

    p.cam_f = cam_f;
    p.cam_c[0] = cam_c[0];
    p.cam_c[1] = cam_c[1];
    p.max_samples = max_samples;
    //printf("[Renderer] max_dist: %ld\n", max_dist);

    p.voxel_dims[0] = in_voxel.size(0);
    p.voxel_dims[1] = in_voxel.size(1);
    p.voxel_dims[2] = in_voxel.size(2);
    p.voxel_strides[0] = in_voxel.stride(0);
    p.voxel_strides[1] = in_voxel.stride(1);
    p.voxel_strides[2] = in_voxel.stride(2);

    //printf("[Renderer] Voxel resolution: %ld, %ld, %ld\n", p.voxel_dims[0], p.voxel_dims[1], p.voxel_dims[2]);

    p.img_dims[0] = img_dims[0];
    p.img_dims[1] = img_dims[1];

    // Create output tensors
    // For Minecraft Seg Mask
    torch::Tensor out_voxel_id = torch::empty({p.img_dims[0], p.img_dims[1], p.max_samples, 1}, torch::TensorOptions().dtype(torch::kInt32).device(device));

    torch::Tensor out_depth;
    // Produce two sets of localcoords, one for entry point, the other one for exit point. They share the same corner_ids.
    out_depth = torch::empty({2, p.img_dims[0], p.img_dims[1], p.max_samples, 1}, torch::TensorOptions().dtype(torch::kFloat32).device(device));

    torch::Tensor out_raydirs = torch::empty({p.img_dims[0], p.img_dims[1], 1, 3}, torch::TensorOptions().dtype(torch::kFloat32).device(device).requires_grad(false));

    const int TILE_DIM = 8;
    dim3 dimGrid((p.img_dims[1]+TILE_DIM-1)/TILE_DIM, (p.img_dims[0]+TILE_DIM-1)/TILE_DIM, 1);
    dim3 dimBlock(TILE_DIM, TILE_DIM, 1);

    ray_voxel_intersection_perspective_kernel<TILE_DIM><<<dimGrid, dimBlock, 0, stream>>>(
        out_voxel_id.data_ptr<int32_t>(), out_depth.data_ptr<float>(), out_raydirs.data_ptr<float>(), in_voxel.data_ptr<int32_t>(), p
    );

    return {out_voxel_id, out_depth, out_raydirs};
}
