#include "hip/hip_runtime.h"
// Copyright (C) 2021 NVIDIA CORPORATION & AFFILIATES.  All rights reserved.
//
// This work is made available under the Nvidia Source Code License-NC.
// To view a copy of this license, check out LICENSE.md

#include <torch/types.h>

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include <time.h>


#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <vector>


#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)


struct PE_Params {
    int ndegrees;
    int pre_size;
    int post_size;
    bool incl_orig;
};

// const int TILE_DIM_X = 16;  // channel dim
// const int TILE_DIM_Y = 64;  // entry dim
// dim3 dimGrid((p.post_size+TILE_DIM_X-1)/TILE_DIM_X, (p.pre_size+TILE_DIM_Y-1)/TILE_DIM_Y, 1);
// dim3 dimBlock(TILE_DIM_X, TILE_DIM_Y, 1);
template <int TILE_DIM_X, int TILE_DIM_Y, int DUP_Y>
__global__ void positional_encoding_kernel(
    float* __restrict__ out_feature,
    const float* __restrict__ in_feature, const PE_Params p) {

    const int idx_feat = blockIdx.x * TILE_DIM_X + threadIdx.x;
    const int idx_entry_base = blockIdx.y * TILE_DIM_Y * DUP_Y + threadIdx.y * DUP_Y;
    if (idx_feat >= p.post_size) {
        return;
    }

    int stride = p.ndegrees*2;
    if (p.incl_orig) {
        stride += 1;
    }

    for (int j=0; j<DUP_Y; j++) {
        int idx_entry = idx_entry_base + j;
        if (idx_entry >= p.pre_size) {
            return;
        }
        float data = in_feature[idx_entry*p.post_size + idx_feat];

        for (int i=0; i<p.ndegrees; i++) {
            float rad = data * HIP_PI_F * exp2f(i);
            //float rad = scalbnf(data * HIP_PI_F, i);
            float sinrad, cosrad;
            sincosf(rad, &sinrad, &cosrad);
            out_feature[idx_entry*p.post_size*stride + i*2*p.post_size + idx_feat] = sinrad;
            out_feature[idx_entry*p.post_size*stride + (i*2+1)*p.post_size + idx_feat] = cosrad;
        }
        if (p.incl_orig) {
            out_feature[idx_entry*p.post_size*stride + (stride-1)*p.post_size + idx_feat] = data;
        }
    }
}

template <int TILE_DIM_X, int TILE_DIM_Y, int DUP_Y>
__global__ void positional_encoding_backward_kernel(
    float* __restrict__ in_feature_grad,
    const float* __restrict__ out_feature_grad, const float* __restrict__ out_feature, const PE_Params p) {

    int idx_feat = blockIdx.x * TILE_DIM_X + threadIdx.x;
    const int idx_entry_base = blockIdx.y * TILE_DIM_Y * DUP_Y + threadIdx.y * DUP_Y;

    if (idx_feat >= p.post_size) {
        return;
    }

    int stride = p.ndegrees*2;
    if (p.incl_orig) {
        stride += 1;
    }

    for (int j=0; j<DUP_Y; j++) {
        int idx_entry = idx_entry_base + j;
        if (idx_entry >= p.pre_size) {
            return;
        }

        float grad = 0.0f;
        for (int i=0; i<p.ndegrees; i++) {
            float grad_t;

            grad_t = out_feature_grad[idx_entry*p.post_size*stride + i*2*p.post_size + idx_feat] *
                out_feature[idx_entry*p.post_size*stride + (i*2+1)*p.post_size + idx_feat];        // cos(x*pi*(2^i))

            grad_t -= out_feature_grad[idx_entry*p.post_size*stride + (i*2+1)*p.post_size + idx_feat] *
                out_feature[idx_entry*p.post_size*stride + (i*2)*p.post_size + idx_feat];        // -sin(x*pi*(2^i))

            grad += grad_t * HIP_PI_F * exp2f(i);
        }
        if (p.incl_orig) {
            grad += out_feature_grad[idx_entry*p.post_size*stride + (stride-1)*p.post_size + idx_feat];
        }

        in_feature_grad[idx_entry*p.post_size + idx_feat] = grad;
    }
}


// Input:
//      in_feature:     float32 [..., N, ...]
//      ndegree:        int32   Degrees of PE encoding
//      dim:            int32   Dimension to concatenate
//      incl_orig:      bool    Whether to include original feature vector or not
// Output:
//      out_feature:     float32 [..., N*ndegree*2+incl_orig, ...]
// std::vector<torch::Tensor>
torch::Tensor positional_encoding_cuda(const torch::Tensor& in_feature, int ndegrees, int dim, bool incl_orig) {
    CHECK_CUDA(in_feature);

    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);
    torch::Device device = in_feature.device();

    assert(in_feature.dtype() == torch::kFloat32);

    // Handle negative index
    if (dim < 0) {
        dim = in_feature.dim() + dim;
    }
    assert(dim >= 0 && dim < in_feature.dim());

    // No need to be contiguous. Input and output has the same memory layout.
    CHECK_CONTIGUOUS(in_feature);

    PE_Params p;
    p.ndegrees = ndegrees;
    p.incl_orig = incl_orig;

    // This only works for contiguous tensors...
    int pre_size = 1;
    int post_size = 1;
    for (int i=0; i<dim; i++) {
        pre_size *= in_feature.size(i);
    }
    for (int i=dim; i<in_feature.dim(); i++) {
        post_size *= in_feature.size(i);
    }
    p.pre_size = pre_size;
    p.post_size = post_size;

    // Calculate output shape
    std::vector<int64_t> out_feature_shape;
    for (int i=0; i<in_feature.dim(); i++) {
        int64_t dim_t = in_feature.size(i);
        if (i == dim) {
            if (incl_orig) {
                dim_t = dim_t*(ndegrees*2+1);
            } else {
                dim_t = dim_t*ndegrees*2;
            }
        }
        out_feature_shape.push_back(dim_t);
    }

    // Always produce contiguous output
    torch::Tensor out_feature = torch::empty(out_feature_shape, torch::TensorOptions().dtype(torch::kFloat32).device(device));

    // Launch CUDA kernel
    // Case 1: Concat at the last dimension (post_size < pre_size)  -->  Each thread handle a single post_size
    // Case 2: Concat at the middle (post_size > pre_size)  -->  Each thread handle
    const int TILE_DIM_X = 16;  // channel dim
    const int TILE_DIM_Y = 64;  // entry dim
    //const int DUP_Y = 4; // Each thread handle multiple entries to save threads
    const int DUP_Y = 8; // DGXA 64 samples per ray @ 256x256
    dim3 dimGrid((p.post_size+TILE_DIM_X-1)/TILE_DIM_X, (p.pre_size+(TILE_DIM_Y*DUP_Y)-1)/(TILE_DIM_Y*DUP_Y), 1);
    dim3 dimBlock(TILE_DIM_X, TILE_DIM_Y, 1);
    positional_encoding_kernel<TILE_DIM_X, TILE_DIM_Y, DUP_Y><<<dimGrid, dimBlock, 0, stream>>>(
        out_feature.data_ptr<float>(),
        in_feature.data_ptr<float>(), p
    );

    THCudaCheck(hipGetLastError());
    return out_feature;
}

//in_feature_grad = voxrender_op.positional_encoding_backward(out_feature_grad, out_feature, ctx.pe_degrees, ctx.dim, ctx.incl_orig);
// Input:
//      out_feature_grad:   float32 [..., N*ndegree*2+incl_orig, ...]
//      out_feature:        float32 [..., N*ndegree*2+incl_orig, ...]
//      ndegrees:           int32   Degrees of PE encoding
//      dim:                int32   Dimension to concatenate
//      incl_orig:          bool    Whether to include original feature vector or not
// Output:
//      in_feature_grad:    float32 [..., N, ...]
// std::vector<torch::Tensor>
torch::Tensor positional_encoding_backward_cuda(const torch::Tensor& out_feature_grad_, const torch::Tensor& out_feature, int ndegrees, int dim, bool incl_orig) {
    CHECK_CUDA(out_feature_grad_);
    CHECK_CUDA(out_feature);

    const torch::Tensor out_feature_grad = out_feature_grad_.contiguous();

    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);
    torch::Device device = out_feature_grad.device();

    assert(out_feature_grad.dtype() == torch::kFloat32);
    assert(out_feature.dtype() == torch::kFloat32);
    assert(out_feature_grad.sizes() == out_feature.sizes());

    // Handle negative index
    if (dim < 0) {
        dim = out_feature.dim() + dim;
    }
    assert(dim >= 0 && dim < out_feature.dim());

    CHECK_CONTIGUOUS(out_feature_grad);
    CHECK_CONTIGUOUS(out_feature);

    PE_Params p;
    p.ndegrees = ndegrees;
    p.incl_orig = incl_orig;

    int expansion_factor = ndegrees*2;
    if (incl_orig) {
        expansion_factor += 1;
    }
    // This only works for contiguous tensors...
    int pre_size = 1;
    int post_size = 1;
    for (int i=0; i<dim; i++) {
        pre_size *= out_feature.size(i);
    }
    for (int i=dim; i<out_feature.dim(); i++) {
        post_size *= out_feature.size(i);
    }
    post_size = post_size / expansion_factor;
    p.pre_size = pre_size;
    p.post_size = post_size;

    // Calculate output shape
    std::vector<int64_t> out_feature_shape;
    for (int i=0; i<out_feature.dim(); i++) {
        int64_t dim_t = out_feature.size(i);
        if (i == dim) {
            dim_t = dim_t / expansion_factor;
        }
        out_feature_shape.push_back(dim_t);
    }

    // Always produce contiguous output
    torch::Tensor in_feature_grad = torch::empty(out_feature_shape, torch::TensorOptions().dtype(torch::kFloat32).device(device));


    // Launch CUDA kernel
    // Case 1: Concat at the last dimension (post_size < pre_size)  -->  Each thread handle a single post_size
    // Case 2: Concat at the middle (post_size > pre_size)  -->  Each thread handle
    const int TILE_DIM_X = 16;  // channel dim
    const int TILE_DIM_Y = 64;  // entry dim
    //const int DUP_Y = 4; // Nothing to amortize
    const int DUP_Y = 8; // DGXA
    dim3 dimGrid((p.post_size+TILE_DIM_X-1)/TILE_DIM_X, (p.pre_size+(TILE_DIM_Y*DUP_Y)-1)/(TILE_DIM_Y*DUP_Y), 1);
    dim3 dimBlock(TILE_DIM_X, TILE_DIM_Y, 1);
    positional_encoding_backward_kernel<TILE_DIM_X, TILE_DIM_Y, DUP_Y><<<dimGrid, dimBlock, 0, stream>>>(
        in_feature_grad.data_ptr<float>(),
        out_feature_grad.data_ptr<float>(), out_feature.data_ptr<float>(), p
    );

    THCudaCheck(hipGetLastError());

    return in_feature_grad;
}
